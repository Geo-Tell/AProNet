#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved.
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCDeviceUtils.cuh>

#include <vector>
#include <iostream>

int const threadsPerBlock = sizeof(unsigned long long) * 8;

__device__ inline float devIoU(float const * const a, float const * const b) {
  float left = max(a[0], b[0]), right = min(a[2], b[2]);
  float top = max(a[1], b[1]), bottom = min(a[3], b[3]);
  float width = max(right - left + 1, 0.f), height = max(bottom - top + 1, 0.f);
  float interS = width * height;
  float Sa = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
  float Sb = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
  return interS / (Sa + Sb - interS);
}

__global__ void nms_kernel(const int n_boxes, const float nms_overlap_thresh,
                           const float *dev_boxes, unsigned long long *dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

  // 这个地方，一般取值是 threadPerBlock
  // 但是最后一个 block 可能存在不足一个 block 的情况, 因此要做一个 min
  const int row_size =
        min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ float block_boxes[threadsPerBlock * 5]; // 这个东西装了一个 block 的 boxes
  // 根据线程 block 内部线程索引 设定该线程所代表的 box
  // NOTE 注意，和 col 相关的是被比的
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 5 + 0] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 0];
    block_boxes[threadIdx.x * 5 + 1] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 1];
    block_boxes[threadIdx.x * 5 + 2] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 2];
    block_boxes[threadIdx.x * 5 + 3] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 3];
    block_boxes[threadIdx.x * 5 + 4] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 4];
  }
  // 进行了同步线程之后, 一个block之内的线程同步
  // 注意 blcok_boxes 是 shared，所以此时一个块的 boxes 全部得到了
  __syncthreads();

  // 这个 if 判断依然是为了解决最后不足 1 个 block 的问题
  // 如果我这个线程所执行的是后面的 padding 部分，那还执行个啥玩意儿？就跳过算了
  // NOTE 注意，和 row 相关的是主动去比的
  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    //                          block 外偏移              block 内偏移

    const float *cur_box = dev_boxes + cur_box_idx * 5;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    // 对角线上的格子，只算一半
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (devIoU(cur_box, block_boxes + i * 5) > nms_overlap_thresh) {
        // 跟 threadIdx 这个 box 的 IoU 大于阈值的所有 boxes 都会被标注为 1
        t |= 1ULL << i;
      }
    }
    const int col_blocks = THCCeilDiv(n_boxes, threadsPerBlock); // col_blocks 个 block 列
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

// boxes is a N x 5 tensor
at::Tensor nms_cuda(const at::Tensor boxes, float nms_overlap_thresh) {
  using scalar_t = float;
  AT_ASSERTM(boxes.type().is_cuda(), "boxes must be a CUDA tensor");
  auto scores = boxes.select(1, 4);
  auto order_t = std::get<1>(scores.sort(0, /* descending=*/true));
  auto boxes_sorted = boxes.index_select(0, order_t);

  int boxes_num = boxes.size(0);

  const int col_blocks = THCCeilDiv(boxes_num, threadsPerBlock);

  scalar_t* boxes_dev = boxes_sorted.data<scalar_t>();

  THCState *state = at::globalContext().lazyInitCUDA(); // TODO replace with getTHCState

  unsigned long long* mask_dev = NULL;
  //THCudaCheck(THCudaMalloc(state, (void**) &mask_dev,
  //                      boxes_num * col_blocks * sizeof(unsigned long long)));

  mask_dev = (unsigned long long*) THCudaMalloc(state, boxes_num * col_blocks * sizeof(unsigned long long));

  dim3 blocks(THCCeilDiv(boxes_num, threadsPerBlock),
              THCCeilDiv(boxes_num, threadsPerBlock));
  dim3 threads(threadsPerBlock);
  nms_kernel<<<blocks, threads>>>(boxes_num,
                                  nms_overlap_thresh,
                                  boxes_dev,
                                  mask_dev);

  std::vector<unsigned long long> mask_host(boxes_num * col_blocks);
  THCudaCheck(hipMemcpy(&mask_host[0],
                        mask_dev,
                        sizeof(unsigned long long) * boxes_num * col_blocks,
                        hipMemcpyDeviceToHost));

  std::vector<unsigned long long> remv(col_blocks);
  memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);

  at::Tensor keep = at::empty({boxes_num}, boxes.options().dtype(at::kLong).device(at::kCPU));
  int64_t* keep_out = keep.data<int64_t>();

  int num_to_keep = 0;
  for (int i = 0; i < boxes_num; i++) {
    // 这个 box 是在哪个 block 里面
    int nblock = i / threadsPerBlock;
    // 这个 box 是在 block 中的第几个
    int inblock = i % threadsPerBlock;

    //    这个东西表示的是一个 block 中所有 boxes 的去除情况，然后使用 1ULL << inblock 来索引一下
    if (!(remv[nblock] & (1ULL << inblock))) {
      keep_out[num_to_keep++] = i; // 将这个 box 标记为保留。在一开始，remv 全都是 0，因此第一个肯定被保留
      // 然后将这个 box 的重叠记录拿出来
      unsigned long long *p = &mask_host[0] + i * col_blocks;
      // 这个 box 的重叠记录是它和其他所有 boxes 的 IoU，这些个 IoU 是分成了很多个 block 的
      // 所以取重叠记录的时候是按着 block 一个一个取的
      // 比如说要查看我和班上每一个人的缘分指数，当我被保留了之后，设置剩下的人是否留下来，这个事情一次做一组，而不是一次一个人
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
    }
  }

  THCudaFree(state, mask_dev);
  // TODO improve this part
  return std::get<0>(order_t.index({
                       keep.narrow(/*dim=*/0, /*start=*/0, /*length=*/num_to_keep).to(
                         order_t.device(), keep.scalar_type())
                     }).sort(0, false));
}
